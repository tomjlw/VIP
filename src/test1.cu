#include <hip/hip_runtime.h>
#include <cstddef>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <chrono> 

using namespace std;
using namespace std::chrono; 

__global__ void filterData(const float *d_data,
                           const float *d_numerator, 
                           float *d_filteredData, 
                           const int numeratorLength,
                           const int filteredDataLength)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < filteredDataLength)
    {   float sum = 0.0f;
        for (int j = 0; j < numeratorLength; j++)
        {
            // The first (numeratorLength-1) elements contain the filter state
            sum += d_numerator[j] * d_data[i + numeratorLength - j - 1];
        }
        d_filteredData[i] = sum;
    }

    //d_filteredData[i] = sum;
}

int main(void)
{
    // (Skipping error checks to make code more readable)
    clock_t start, end;
    unsigned long micros = 0;
    int dataLength = 1024;
    int filteredDataLength = 1024;
    int numeratorLength= 1024;

    // Pointers to data, filtered data and filter coefficients
    // (Skipping how these are read into the arrays)
    float *h_data = new float[dataLength];
    float *h_filteredData = new float[filteredDataLength];
    float *h_filter = new float[numeratorLength];
      
    for (int i=0;i<dataLength;i++){
    h_data[i] = (float)rand()/(float)(RAND_MAX/2);
    }
    for (int j=0;j<numeratorLength;j++){ 
    h_filter[j] = (float)rand()/(float)(RAND_MAX/0.5);
    }
    
    // Create device pointers
    float *d_data = nullptr;
    hipMalloc((void **)&d_data, dataLength * sizeof(float));

    float *d_numerator = nullptr;
    hipMalloc((void **)&d_numerator, numeratorLength * sizeof(float));

    float *d_filteredData = nullptr;
    hipMalloc((void **)&d_filteredData, filteredDataLength * sizeof(float));


    // Copy data to device
    hipMemcpy(d_data, h_data, dataLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_numerator, h_filter, numeratorLength * sizeof(float), hipMemcpyHostToDevice);  

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (filteredDataLength + threadsPerBlock - 1) / threadsPerBlock;
    start = clock();
    filterData<<<blocksPerGrid,threadsPerBlock>>>(d_data, d_numerator, d_filteredData, numeratorLength, filteredDataLength);
    end = clock();
    micros = end - start;
    cout<<micros;

    // Copy results to host
    hipMemcpy(h_filteredData, d_filteredData, filteredDataLength * sizeof(float), hipMemcpyDeviceToHost);
    //for (int i=0;i<filteredDataLength;i++){
	//printf("%lf\n",h_filteredData[i]);}
    // Clean up
    hipFree(d_data);
    hipFree(d_numerator);
    hipFree(d_filteredData);

    // Do stuff with h_filteredData...

    // Clean up some more
    delete [] h_data;
    delete [] h_filteredData;
    delete [] h_filter;
}
