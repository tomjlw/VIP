#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stddef.h>
#include <cstddef>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <chrono> 
#include "read.h"

using namespace std;
using namespace std::chrono; 

#define Lh 65 //filter length
#define Lx 1201 //input signal length

__global__ void filterData(const float *d_data,
                           const float *d_numerator, 
                           float *d_filteredData, 
                           const int numeratorLength,
                           const int filteredDataLength)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   float sum = 0;

    if (i < filteredDataLength)
    {
        for (int j = 0; j < numeratorLength; j++)
        {
            sum += d_numerator[j] * d_data[i + numeratorLength - j - 1];
        }
    }

    d_filteredData[i] = sum;
}


int main(void)
{   float z[10000], b[10000]; 
    read(z, b);
   
    // (Skipping error checks to make code more readable)
    clock_t start, end;
    unsigned long micros = 0;
    int dataLength = Lx;
    int filteredDataLength = Lx;
    int numeratorLength= Lh;
    
    // Pointers to data, filtered data and filter coefficients
    // (Skipping how these are read into the arrays)
    float *h_data = new float[dataLength];
    float *h_filteredData = new float[filteredDataLength];
    float *h_filter = new float[numeratorLength];
      
    for (int i=0;i<dataLength;i++){
    h_data[i] = z[i];
    }
    for (int j=0;j<numeratorLength;j++){
    h_filter[j] = b[j];
    }

    // Create device pointers
    float *d_data = nullptr;
    hipMalloc((void **)&d_data, dataLength * sizeof(float));

    float *d_numerator = nullptr;
    hipMalloc((void **)&d_numerator, numeratorLength * sizeof(float));

    float *d_filteredData = nullptr;
    hipMalloc((void **)&d_filteredData, filteredDataLength * sizeof(float));


    // Copy data to device
    hipMemcpy(d_data, h_data, dataLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_numerator, h_filter, numeratorLength * sizeof(float), hipMemcpyHostToDevice);  

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (filteredDataLength + threadsPerBlock - 1) / threadsPerBlock;
    start = clock();
    filterData<<<blocksPerGrid,threadsPerBlock>>>(d_data, d_numerator, d_filteredData, numeratorLength, filteredDataLength);
    end = clock();
    micros = end - start;
    //cout<<micros;

    // Copy results to host
    hipMemcpy(h_filteredData, d_filteredData, filteredDataLength * sizeof(float),  hipMemcpyDeviceToHost);
    for (int i=0;i<filteredDataLength;i++){
    printf("%lf\n",h_filteredData[i]);}
    // Clean up
    hipFree(d_data);
    hipFree(d_numerator);
    hipFree(d_filteredData);

    // Do stuff with h_filteredData...

    // Clean up some more
    delete [] h_data;
    delete [] h_filteredData;
    delete [] h_filter;
}
